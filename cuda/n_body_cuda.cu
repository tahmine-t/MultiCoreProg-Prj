#include "hip/hip_runtime.h"
/* Sequential version of N body simulation */

#include "Dataset/NBody-600.h"
#include "VectorMath.h"
#include "CycleTimer.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>

using namespace std;

int COMPUTATION_STEP = 100;


// Compute forces on each body with time step

// Physics

__device__
void updateAcceleration(int bodyIndex, 
        Position3D *d_pos,
        Acceleration3D *d_acc,
        Mass *d_mass) 
{
   
  Force3D netForce = { 0, 0, 0 };

  for( int i = 0; i < BODY_COUNT; i++ ) 
  {
    if( i == bodyIndex ) 
    {
      continue;
    }

    Force3D vectorForceToOther = {0, 0, 0};
    Force scalarForceBetween = forceNewtonianGravity3D(
                                  d_mass[bodyIndex],
                                  d_mass[i],
                                  d_pos[bodyIndex],
                                  d_pos[i]);
    direction( 
      d_pos[bodyIndex],
      d_pos[i],
      vectorForceToOther);

    vectorForceToOther.x *= scalarForceBetween;
    vectorForceToOther.y *= scalarForceBetween;
    vectorForceToOther.z *= scalarForceBetween;
    netForce.x += vectorForceToOther.x;
    netForce.y += vectorForceToOther.y;
    netForce.z += vectorForceToOther.z;
  }

  d_acc[bodyIndex] = computeAccel3D(d_mass[bodyIndex], netForce);
}

__device__
void updateVelocity(
      int bodyIndex, 
      float deltaT, 
      Acceleration3D *d_acc,
      Velocity3D *d_vel
      ) 
{
  d_vel[bodyIndex] = computeVelo3D(
                                d_acc[bodyIndex],
                                d_vel[bodyIndex],
                                deltaT);
}

__device__
void updatePosition(
        int bodyIndex, 
        float deltaT,
        Velocity3D *d_vel,
        Position3D *d_pos) 
{

  d_pos[bodyIndex] = computePos3D( 
                              d_vel[bodyIndex],
                              d_pos[bodyIndex],
                              deltaT);
}

__global__
void updatePhysics(
        int bodies,
        float deltaT, 
        Position3D *d_pos,
        Velocity3D *d_vel,
        Acceleration3D *d_acc,
        Mass *d_mass)
{
  
  int i = blockIdx.x;
  int j = threadIdx.x;

  int body_id = (i * j) + j;

  if(body_id > bodies)
    return;
  
  updateAcceleration(body_id, d_pos, d_acc, d_mass);
  updateVelocity(body_id, deltaT, d_acc, d_vel);
  updatePosition(body_id, deltaT, d_vel, d_pos);
}

void display_bodies(ofstream &outfile)
{
  for (int i = 0; i < BODY_COUNT; i++){

    outfile << "Body " << i + 1 << ":" << endl;
    outfile << "Mass: " << nBodyMass[i] << endl;
    outfile << "Position(x, y, z): "
            << nBodyPosition[i].x << " " << nBodyPosition[i].y << " " << nBodyPosition[i].z << endl;
    outfile << "Velocity(x, y, z): "
            << nBodyVelocity[i].x << " " << nBodyVelocity[i].y << " " << nBodyVelocity[i].z << endl;
    outfile << "Acceleration(x, y, z): "
            << nBodyAcceleration[i].x << " " << nBodyAcceleration[i].y << " " << nBodyAcceleration[i].z << endl
            << endl;
  }
}


void compute(){

  int BYTES_SIZE_VECTOR = BODY_COUNT * sizeof(Vector3D);
  int BYTES_SIZE_SCALAR = BODY_COUNT * sizeof(Scalar);

  //Initializing Velocities of N bodies in GPU
  Velocity3D *h_vel = nBodyVelocity;
  Velocity3D *d_vel;
  hipMalloc((void**) &d_vel, BYTES_SIZE_VECTOR);
  hipMemcpy(d_vel, h_vel, BYTES_SIZE_VECTOR, hipMemcpyHostToDevice);

  //Initializing acceleration of N bodies in GPU
  Acceleration3D *h_acc = nBodyAcceleration;
  Acceleration3D *d_acc;
  hipMalloc((void**) &d_acc, BYTES_SIZE_VECTOR);
  hipMemcpy(d_acc, h_acc, BYTES_SIZE_VECTOR, hipMemcpyHostToDevice);
  
  //Initializing Mass of N bodies in GPU
  Mass *h_mass = nBodyMass;
  Mass *d_mass;
  hipMalloc((void**) &d_mass, BYTES_SIZE_SCALAR);
  hipMemcpy(d_mass, h_mass, BYTES_SIZE_SCALAR, hipMemcpyHostToDevice);

  //Initializing Positions of N bodies in GPU
  Position3D *h_pos = nBodyPosition;
  Position3D *d_pos;
  hipMalloc((void**) &d_pos, BYTES_SIZE_VECTOR);
  hipMemcpy(d_pos, h_pos, BYTES_SIZE_VECTOR, hipMemcpyHostToDevice);


  for (int i = 0; i < COMPUTATION_STEP; ++i){
    updatePhysics<<<(BODY_COUNT/16) + 1, 16>>>(BODY_COUNT, (float)(i * 100), d_pos, d_vel, d_acc, d_mass);
  }
    

  hipMemcpy(h_pos, d_pos, BYTES_SIZE_VECTOR, hipMemcpyDeviceToHost);
  hipMemcpy(h_vel, d_vel, BYTES_SIZE_VECTOR, hipMemcpyDeviceToHost);
  hipMemcpy(h_acc, d_acc, BYTES_SIZE_VECTOR, hipMemcpyDeviceToHost);
  hipFree(d_pos);
  hipFree(d_vel);
  hipFree(d_acc);
  hipFree(d_mass);

}


int main(){
  double start, end, min = 1e30;
  start = CycleTimer::currentSeconds();

  compute();
  
  end = CycleTimer::currentSeconds();
  min = std::min(min, end - start);

  // Write Results in output file
  ofstream file_name("nbody_parallel.txt");
  if (!file_name.is_open())
  {
    cerr << "Error opening file!" << endl;
    return 0;
  }
  file_name << "Body Count: " << BODY_COUNT << endl;
  file_name << "Total Time: " << min << " seconds" << endl
            << endl;
  display_bodies(file_name);
  file_name.close();
  
  printf("Time: %f\n", min);
}