#include "hip/hip_runtime.h"
/* Sequential version of N body simulation */
/* Author: Raghav Pandya */

#include "NBody.h"
#include "VectorMath.h"
#include "CycleTimer.h"
#include <hip/hip_runtime.h>

using namespace std;

// Compute forces on each body with time step

// Physics

__device__
void updateAcceleration(int bodyIndex, 
        Position3D *d_pos,
        Acceleration3D *d_acc,
        Mass *d_mass) 
{
   
  Force3D netForce = { 0, 0, 0 };

  for( int i = 0; i < BODY_COUNT; i++ ) 
  {
    if( i == bodyIndex ) 
    {
      continue;
    }

    Force3D vectorForceToOther = {0, 0, 0};
    Force scalarForceBetween = forceNewtonianGravity3D(
                                  d_mass[bodyIndex],
                                  d_mass[i],
                                  d_pos[bodyIndex],
                                  d_pos[i]);
    direction( 
      d_pos[bodyIndex],
      d_pos[i],
      vectorForceToOther);

    vectorForceToOther.x *= scalarForceBetween;
    vectorForceToOther.y *= scalarForceBetween;
    vectorForceToOther.z *= scalarForceBetween;
    netForce.x += vectorForceToOther.x;
    netForce.y += vectorForceToOther.y;
    netForce.z += vectorForceToOther.z;
  }

  d_acc[bodyIndex] = computeAccel3D(d_mass[bodyIndex], netForce);
}

__device__
void updateVelocity(
      int bodyIndex, 
      float deltaT, 
      Acceleration3D *d_acc,
      Velocity3D *d_vel
      ) 
{
  d_vel[bodyIndex] = computeVelo3D(
                                d_acc[bodyIndex],
                                d_vel[bodyIndex],
                                deltaT);
}

__device__
void updatePosition(
        int bodyIndex, 
        float deltaT,
        Velocity3D *d_vel,
        Position3D *d_pos) 
{

  d_pos[bodyIndex] = computePos3D( 
                              d_vel[bodyIndex],
                              d_pos[bodyIndex],
                              deltaT);
}

__global__
void updatePhysics(
        int bodies,
        float deltaT, 
        Position3D *d_pos,
        Velocity3D *d_vel,
        Acceleration3D *d_acc,
        Mass *d_mass)
{
  
  int i = blockIdx.x;
  int j = threadIdx.x;

  int body_id = (i * j) + j;

  if(body_id > bodies)
    return;
  
  updateAcceleration(body_id, d_pos, d_acc, d_mass);
  updateVelocity(body_id, deltaT, d_acc, d_vel);
  updatePosition(body_id, deltaT, d_vel, d_pos);
}


void compute() 
{
  double start, end, min = 1e30;

  int BYTES_SIZE_VECTOR = BODY_COUNT * sizeof(Vector3D);
  int BYTES_SIZE_SCALAR = BODY_COUNT * sizeof(Scalar);

  //Initializing Velocities of N bodies in GPU
  Velocity3D *h_vel = nBodyVelocity;
  Velocity3D *d_vel;
  hipMalloc((void**) &d_vel, BYTES_SIZE_VECTOR);
  hipMemcpy(d_vel, h_vel, BYTES_SIZE_VECTOR, hipMemcpyHostToDevice);

  //Initializing acceleration of N bodies in GPU
  Acceleration3D *h_acc = nBodyAcceleration;
  Acceleration3D *d_acc;
  hipMalloc((void**) &d_acc, BYTES_SIZE_VECTOR);
  hipMemcpy(d_acc, h_acc, BYTES_SIZE_VECTOR, hipMemcpyHostToDevice);
  
  //Initializing Mass of N bodies in GPU
  Mass *h_mass = nBodyMass;
  Mass *d_mass;
  hipMalloc((void**) &d_mass, BYTES_SIZE_SCALAR);
  hipMemcpy(d_mass, h_mass, BYTES_SIZE_SCALAR, hipMemcpyHostToDevice);

    //Initializing Positions of N bodies in GPU
  Position3D *h_pos = nBodyPosition;
  Position3D *d_pos;
  hipMalloc((void**) &d_pos, BYTES_SIZE_VECTOR);
  hipMemcpy(d_pos, h_pos, BYTES_SIZE_VECTOR, hipMemcpyHostToDevice);


  for (int j = 0; j < 3; ++j)
  {
  start = CycleTimer::currentSeconds(); 
  for (int i = 0; i < 10000; ++i)
  {
    updatePhysics<<<(BODY_COUNT/16) + 1, 16>>>(BODY_COUNT, (float)(i * 100), d_pos, d_vel, d_acc, d_mass);
  }

  end = CycleTimer::currentSeconds();
  min = std::min(min, end - start);
    
  }
  hipMemcpy(h_pos, d_pos, BYTES_SIZE_VECTOR, hipMemcpyDeviceToHost);
  hipMemcpy(h_vel, d_vel, BYTES_SIZE_VECTOR, hipMemcpyDeviceToHost);
  hipMemcpy(h_acc, d_acc, BYTES_SIZE_VECTOR, hipMemcpyDeviceToHost);
  hipFree(d_pos);
  hipFree(d_vel);
  hipFree(d_acc);
  hipFree(d_mass);

  printf("Time Taken by CUDA implementation: %f ms\n", (min)*1000);
}


int main() 
{
  compute();
}